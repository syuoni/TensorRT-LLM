#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/cuteDslKernels/moeUtils.h"
#include "tensorrt_llm/kernels/quantization.cuh"

#include <hip/hip_fp4.h>
#include <cute/numeric/numeric_types.hpp>

namespace tensorrt_llm::kernels::cute_dsl
{
namespace
{
using ElemCopyType = uint4;
using SFCopyType = uint32_t;

template <typename T>
auto constexpr bitsPerElem()
{
#ifdef ENABLE_FP4
    return std::is_same_v<T, __hip_fp4_e2m1> ? 4 : cute::sizeof_bits_v<T>;
#else
    return cute::sizeof_bits_v<T>;
#endif
}

template <typename T>
auto constexpr elemPerCopy()
{
    return bitsPerElem<ElemCopyType>() / bitsPerElem<T>();
}

template <typename T>
auto constexpr sfElemPerCopy()
{
    return bitsPerElem<SFCopyType>() / bitsPerElem<T>();
}
} // namespace

template <typename InputType, typename SFType, int32_t kSFVecSize, int32_t kThreadsPerBlock>
__global__ void moePermuteKernel(InputType const* input, InputType* permuted_output, SFType const* input_sf,
    SFType* permuted_sf, int32_t const* permuted_idx_to_expanded_idx, int32_t const* num_non_exiting_tiles,
    int32_t const hidden_size, int32_t const top_k, int32_t const tile_size)
{
    int32_t constexpr kElemPerCopy = elemPerCopy<InputType>();
    int32_t constexpr kSFElemPerCopy = sfElemPerCopy<SFType>();

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    int32_t const num_tokens = num_non_exiting_tiles[0] * tile_size;
    for (int32_t permuted_idx = blockIdx.x; permuted_idx < num_tokens; permuted_idx += gridDim.x)
    {
        int32_t const expanded_idx = permuted_idx_to_expanded_idx[permuted_idx];
        if (expanded_idx < 0)
        {
            continue;
        }
        int32_t const token_idx = expanded_idx / top_k;

        auto const* src_ptr = reinterpret_cast<ElemCopyType const*>(input) + token_idx * hidden_size / kElemPerCopy;
        auto* dst_ptr = reinterpret_cast<ElemCopyType*>(permuted_output) + permuted_idx * hidden_size / kElemPerCopy;
        for (int32_t i = threadIdx.x; i < hidden_size / kElemPerCopy; i += kThreadsPerBlock)
        {
            dst_ptr[i] = src_ptr[i];
        }

#ifdef ENABLE_FP4
        if constexpr (std::is_same_v<InputType, __hip_fp4_e2m1>)
        {
            int32_t const sf_hidden_size = hidden_size / kSFVecSize;
            auto const* sf_src_ptr = reinterpret_cast<SFCopyType const*>(input_sf);
            auto* sf_dst_ptr = reinterpret_cast<SFCopyType*>(permuted_sf);
            for (int32_t i = threadIdx.x; i < sf_hidden_size / kSFElemPerCopy; i += kThreadsPerBlock)
            {
                // input_sf is not swizzled, while permuted_sf is swizzled.
                int32_t const src_offset = token_idx * sf_hidden_size / kSFElemPerCopy + i;
                int32_t const dst_offset = get_sf_out_offset_128x4(/* batchIdx= */ std::nullopt, permuted_idx,
                                               i * kSFElemPerCopy, /* numRows= */ std::nullopt, sf_hidden_size)
                    / kSFElemPerCopy;

                sf_dst_ptr[dst_offset] = sf_src_ptr[src_offset];
            }
        }
#endif
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename InputType, typename SFType>
void moePermute(InputType const* input, InputType* permuted_output, SFType const* input_sf, SFType* permuted_sf,
    int32_t const* permuted_idx_to_expanded_idx, int32_t const* num_non_exiting_tiles, int32_t const hidden_size,
    int32_t const top_k, int32_t const tile_size, hipStream_t stream)
{
    int32_t constexpr kThreadsPerBlock = 256;
    int32_t constexpr kSFVecSize = 16;
    int32_t constexpr kElemPerCopy = elemPerCopy<InputType>();
    TLLM_CHECK_WITH_INFO(hidden_size % kElemPerCopy == 0, "hidden_size must be divisible by %d.", kElemPerCopy);

#ifdef ENABLE_FP4
    if constexpr (std::is_same_v<InputType, __hip_fp4_e2m1>)
    {
        int32_t constexpr kSFElemPerCopy = sfElemPerCopy<SFType>();
        TLLM_CHECK_WITH_INFO(hidden_size % (kSFVecSize * kSFElemPerCopy) == 0, "hidden_size must be divisible by %d.",
            kSFVecSize * kSFElemPerCopy);
        TLLM_CHECK_WITH_INFO(input_sf != nullptr, "input_sf is required for NVFP4.");
        TLLM_CHECK_WITH_INFO(permuted_sf != nullptr, "permuted_sf is required for NVFP4.");
    }
#endif

    static int64_t const smCount = tensorrt_llm::common::getMultiProcessorCount();
    int32_t const blocks = smCount;
    int32_t const threads = kThreadsPerBlock;

    auto kernel = &moePermuteKernel<InputType, SFType, kSFVecSize, kThreadsPerBlock>;

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, kernel, input, permuted_output, input_sf, permuted_sf, permuted_idx_to_expanded_idx,
        num_non_exiting_tiles, hidden_size, top_k, tile_size);
}

#define INSTANTIATE_MOE_PERMUTE(InputType, SFType)                                                                     \
    template void moePermute<InputType, SFType>(InputType const* input, InputType* permuted_output,                    \
        SFType const* input_sf, SFType* permuted_sf, int32_t const* permuted_idx_to_expanded_idx,                      \
        int32_t const* num_non_exiting_tiles, int32_t const hidden_size, int32_t const top_k, int32_t const tile_size, \
        hipStream_t stream)

INSTANTIATE_MOE_PERMUTE(half, uint8_t);
#ifdef ENABLE_BF16
INSTANTIATE_MOE_PERMUTE(__hip_bfloat16, uint8_t);
#endif
#ifdef ENABLE_FP8
INSTANTIATE_MOE_PERMUTE(__hip_fp8_e4m3_fnuz, uint8_t);
#endif
#ifdef ENABLE_FP4
INSTANTIATE_MOE_PERMUTE(__hip_fp4_e2m1, uint8_t);
#endif
#undef INSTANTIATE_MOE_PERMUTE

template <typename InputType, typename TopKScaleType, int32_t kThreadsPerBlock>
__global__ void moeUnpermuteKernel(InputType const* permuted_input, InputType* output,
    int32_t const* expanded_idx_to_permuted_idx, TopKScaleType const* topk_scales, int32_t const hidden_size,
    int32_t const top_k)
{
    using AccumType = float;
    int32_t constexpr kElemPerCopy = elemPerCopy<InputType>();
    InputType rmem[kElemPerCopy];
    AccumType rmemAccum[kElemPerCopy];

    int32_t const token_idx = blockIdx.x;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    auto* dst_ptr = reinterpret_cast<ElemCopyType*>(output) + token_idx * hidden_size / kElemPerCopy;
    for (int32_t i = threadIdx.x; i < hidden_size / kElemPerCopy; i += kThreadsPerBlock)
    {
#pragma unroll
        for (int32_t j = 0; j < kElemPerCopy; j++)
        {
            rmemAccum[j] = 0;
        }
        for (int32_t k = 0; k < top_k; k++)
        {
            int32_t const permuted_idx = expanded_idx_to_permuted_idx[token_idx * top_k + k];
            TopKScaleType const scale = topk_scales[token_idx * top_k + k];
            if (permuted_idx < 0)
            {
                continue;
            }
            auto const* src_ptr
                = reinterpret_cast<ElemCopyType const*>(permuted_input) + permuted_idx * hidden_size / kElemPerCopy;
            *reinterpret_cast<ElemCopyType*>(rmem) = src_ptr[i];

#pragma unroll
            for (int32_t j = 0; j < kElemPerCopy; j++)
            {
                rmemAccum[j] += static_cast<AccumType>(rmem[j]) * static_cast<AccumType>(scale);
            }
        }
#pragma unroll
        for (int32_t j = 0; j < kElemPerCopy; j++)
        {
            rmem[j] = static_cast<InputType>(rmemAccum[j]);
        }
        dst_ptr[i] = *reinterpret_cast<ElemCopyType*>(rmem);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename InputType, typename TopKScaleType>
void moeUnpermute(InputType const* permuted_input, InputType* output, int32_t const* expanded_idx_to_permuted_idx,
    TopKScaleType const* topk_scales, int32_t const num_tokens, int32_t const hidden_size, int32_t const top_k,
    hipStream_t stream)
{
    int32_t constexpr kThreadsPerBlock = 256;
    int32_t constexpr kElemPerCopy = elemPerCopy<InputType>();
    TLLM_CHECK_WITH_INFO(hidden_size % kElemPerCopy == 0, "hidden_size must be divisible by %d.", kElemPerCopy);

    int32_t const blocks = num_tokens;
    int32_t const threads = kThreadsPerBlock;

    auto kernel = &moeUnpermuteKernel<InputType, TopKScaleType, kThreadsPerBlock>;

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(
        &config, kernel, permuted_input, output, expanded_idx_to_permuted_idx, topk_scales, hidden_size, top_k);
}

#define INSTANTIATE_MOE_UNPERMUTE(InputType, TopKScaleType)                                                            \
    template void moeUnpermute<InputType>(InputType const* permuted_input, InputType* output,                          \
        int32_t const* expanded_idx_to_permuted_idx, TopKScaleType const* topk_scales, int32_t const num_tokens,       \
        int32_t const hidden_size, int32_t const top_k, hipStream_t stream)

INSTANTIATE_MOE_UNPERMUTE(half, float);
INSTANTIATE_MOE_UNPERMUTE(half, half);
#ifdef ENABLE_BF16
INSTANTIATE_MOE_UNPERMUTE(__hip_bfloat16, float);
INSTANTIATE_MOE_UNPERMUTE(__hip_bfloat16, __hip_bfloat16);
#endif
#undef INSTANTIATE_MOE_UNPERMUTE

} // namespace tensorrt_llm::kernels::cute_dsl
